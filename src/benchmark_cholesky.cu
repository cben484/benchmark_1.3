#include "../include/check_device.cuh"
#include "../include/cuda_utils_check.hpp"
#include "../include/macro.hpp"
#include "hipblas.h"
#include <cstddef>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <iomanip> // std::setprecision
#include <iostream>
#include <istream>
#include <iterator>
#include <ostream>
#include <string>

int curandgenerate_gemm(double *matrx, int m, int n, unsigned long long seed);
int validate_ss_D(hipblasHandle_t handle, double const *matrix, double *result,
                  double const *origin, int N);
int print_matrix_rowmajor(double *matrix, int m, int n);
template <typename T> int print_matrix_colmajor(T *matrix, int m, int n);
int validate_ss_S(hipblasHandle_t handle, float const *matrix, float *result,
                  float const *origin, int N);

int main(int argc, char *argv[]) {

  std::cout << "*************************************************" << std::endl;
  std::cout << "探测设备......" << std::endl;
  CHECK_Device(&argv[0]);

  std::cout << "*************************************************" << std::endl;
  std::cout << "参数总数：" << std::endl;
  std::cout << argc << std::endl;
  std::cout << "参数检查：" << std::endl;
  std::cout << argv[0] << std::endl;
  std::cout << argv[1] << std::endl;
  std::cout << "参数检查完毕" << std::endl;

  std::cout << "*************************************************" << std::endl;

  std::cout << "此方阵的参数为：" << argv[1] << " x " << argv[1] << std::endl;
  std::cout << "*************************************************" << std::endl;
  std::cout << "开始 " << argv[1] << " " << "x" << " " << argv[1]
            << " 规模的 Double Cholesky 分解" << std::endl;

  // 输入的矩阵的规模
  int INPUTN = std::stoi(argv[1]);

  hipsolverHandle_t cusolver_handle;
  hipblasHandle_t cublas_handle;
  CHECK_Cusolver(hipsolverDnCreate(&cusolver_handle));
  CHECK_Cublas(hipblasCreate(&cublas_handle));
  // Double precision
  double *DA;
  double *DB;
  int DLwork;
  int *DdevInfo;
  double *DWorkspace;
  double Dalpha = 1.0;
  double Dbeta = 0.0;
  double temp = 1.0;
  float one_three = 1.0 / 3.0;

  // 创建显示stream和event（目前此代码中是单stream，和隐式一般无二）
  hipStream_t stream;
  CHECK_Runtime(hipStreamCreate(&stream));
  hipEvent_t start, stop;
  if (hipEventCreate(&start) != hipSuccess) {
    printf("Failed to create start event\n");
    return EXIT_SUCCESS;
  }

  if (hipEventCreate(&stop) != hipSuccess) {
    printf("Failed to create stop event\n");
    CHECK_Runtime(hipEventDestroy(start));
    return EXIT_SUCCESS;
  }

  // 初始化操作,使用hiprand为DA随机初始化赋值
  CHECK_Runtime(hipMalloc((void **)&DA, sizeof(double) * INPUTN * INPUTN));
  if (DA == nullptr) {
    fprintf(stderr, "Memory allocation failed for DA\n");
    exit(EXIT_FAILURE);
  }
  CHECK_Runtime(hipMalloc((void **)&DB, sizeof(double) * INPUTN * INPUTN));
  if (DB == nullptr) {
    fprintf(stderr, "Memory allocation failed for DB\n");
    exit(EXIT_FAILURE);
  }

  hiprandGenerator_t gen;
  CHECK_Curand(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  CHECK_Curand(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
  CHECK_Curand(hiprandGenerateUniformDouble(gen, DB, INPUTN * INPUTN));

  // 原本是查看生成的随机矩阵DB：
  double *h_DB = new double[INPUTN * INPUTN];
  CHECK_Runtime(hipMemcpy(h_DB, DB, sizeof(double) * INPUTN * INPUTN,
                           hipMemcpyDeviceToHost));

  // DB+=I，提高DB的秩，一定程度上确保DB为满秩矩阵
  for (int i = 0; i < INPUTN; ++i) {
    h_DB[i * INPUTN + i] += temp;
  }
  CHECK_Runtime(hipMemcpy(DB, h_DB, sizeof(double) * INPUTN * INPUTN,
                           hipMemcpyHostToDevice));

  // DA=DB^T*DB
  CHECK_Cublas(hipblasDgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, INPUTN,
                           INPUTN, INPUTN, &Dalpha, DB, INPUTN, DB, INPUTN,
                           &Dbeta, DA, INPUTN));

  CHECK_Runtime(hipMemcpy(h_DB, DA, sizeof(double) * INPUTN * INPUTN,
                           hipMemcpyDeviceToHost));

  // 开始前先给GPU热身
  for (size_t i{0}; i < warmups; ++i) {
    CHECK_Cublas(hipblasDgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, INPUTN,
                             INPUTN, INPUTN, &Dalpha, DB, INPUTN, DB, INPUTN,
                             &Dbeta, DB, INPUTN));
  }

  // 要为A分配内存，但是不必初始化，分配内存主要是因为buffer函数需要一个指针参数，并且buffer函数会以此与lda、n做验证，从而确定需要的Lwork

  // buffer之前检查一下DA的模样，原本是查看DA
  CHECK_Runtime(hipMemcpy(h_DB, DA, sizeof(double) * INPUTN * INPUTN,
                           hipMemcpyDeviceToHost));

  CHECK_Cusolver(hipsolverDnDpotrf_bufferSize(
      cusolver_handle, HIPBLAS_FILL_MODE_LOWER, INPUTN, DA, INPUTN, &DLwork));
  // buffer之后检查一下DA的模样，原本是查看DA
  CHECK_Runtime(hipMemcpy(h_DB, DA, sizeof(double) * INPUTN * INPUTN,
                           hipMemcpyDeviceToHost));

  CHECK_Runtime(hipMalloc((void **)&DWorkspace, DLwork * sizeof(double)));

  if (DWorkspace == nullptr) {
    fprintf(stderr, "Memory allocation failed for DWorkspace\n");
    exit(EXIT_FAILURE);
  }

  CHECK_Runtime(hipMalloc((void **)&DdevInfo, sizeof(int)));
  // Dportf之前检查一下DA的模样，原本是查看DA
  double *h_DB_origin;
  h_DB_origin = (double *)malloc(sizeof(double) * INPUTN * INPUTN);
  CHECK_Runtime(hipMemcpy(h_DB_origin, DA, sizeof(double) * INPUTN * INPUTN,
                           hipMemcpyDeviceToHost));

  // 记录start
  CHECK_Runtime(hipEventRecord(start, stream));
  // 要为A和Workspace分配空间，以及初始化A，对A进行potrf

  CHECK_Cusolver(hipsolverDnDpotrf(cusolver_handle, HIPBLAS_FILL_MODE_LOWER,
                                  INPUTN, DA, INPUTN, DWorkspace, DLwork,
                                  DdevInfo));
  // 记录stop
  CHECK_Runtime(hipEventRecord(stop, stream));
  // 同步
  CHECK_Runtime(hipEventSynchronize(stop));
  CHECK_Runtime(hipStreamSynchronize(stream));

  // Dportf之后检查一下DA的模样，原本是查看一下DA
  CHECK_Runtime(hipMemcpy(h_DB, DA, sizeof(double) * INPUTN * INPUTN,
                           hipMemcpyDeviceToHost));

  // 作差求elapse
  float DelapsedTime;
  CHECK_Runtime(hipEventElapsedTime(&DelapsedTime, start, stop));
  // 计算latency
  float const Dlatency{DelapsedTime};
  // 计算TFLOPS
  float const Dtflops{(one_three * INPUTN * INPUTN * INPUTN) /
                      ((Dlatency * 1e-3f) * 1e12f)};
  std::cout << "*************************************************" << std::endl;
  // 输出TFLOPS
  std::cout << "双精度Cholesky分解的TFLOPS: " << Dtflops << " TFLOPS"
            << std::endl;
  // std::cout << "Dpotrf Effective TFLOPS:" << Dtflops << "TFLOPS" <<
  // std::endl; 输出elapse
  printf("\n双精度Cholesky分解的执行时间 : %fms   %fs\n", DelapsedTime,
         DelapsedTime / 1000);
  std::cout << "*************************************************" << std::endl;
  // printf("\n Dpotrf execution time: %fms   %fs\n", DelapsedTime,
  //        DelapsedTime / 1000);

  // validate之前检查一下DA的模样，原本是检查一下DA
  CHECK_Runtime(hipMemcpy(h_DB, DA, sizeof(double) * INPUTN * INPUTN,
                           hipMemcpyDeviceToHost));

  double result;
  // 这里传进去的origin要直接放到memcpy里面作为src以device2host的方式，所以这里的origin需要一个device的版本，但是h_DB是host的，所以一直会报错，解决方案：memcpy
  double *h2d_DB;
  CHECK_Runtime(hipMalloc((void **)&h2d_DB, sizeof(double) * INPUTN * INPUTN));
  CHECK_Runtime(hipMemcpy(h2d_DB, h_DB_origin,
                           sizeof(double) * INPUTN * INPUTN,
                           hipMemcpyHostToDevice));
  std::cout << "双精度验证开始" << std::endl;
  validate_ss_D(cublas_handle, DA, &result, h2d_DB, INPUTN);

  std::cout << std::fixed << std::setprecision(20);
  std::cout << "the validate value is:" << result << std::endl;
  std::cout << "双精度验证完毕" << std::endl;
  std::cout << "*************************************************" << std::endl;
  std::cout << std::endl;
  std::cout << std::endl;

  // Single precision
  float *SB;
  float *SA;
  int SLwork;
  int *SdevInfo;
  float *SWorkspace;
  float Salpha = 1.0;
  float Sbeta = 0.0;
  float *stemp;

  stemp = (float *)malloc(sizeof(float) * INPUTN * INPUTN);
  CHECK_Runtime(hipMalloc((void **)&SA, sizeof(float) * INPUTN * INPUTN));
  if (SA == nullptr) {
    fprintf(stderr, "Memory allocation failed for SA\n");
    exit(EXIT_FAILURE);
  }
  CHECK_Runtime(hipMalloc((void **)&SB, sizeof(float) * INPUTN * INPUTN));
  if (SB == nullptr) {
    fprintf(stderr, "Memory allocation failed for SB\n");
    exit(EXIT_FAILURE);
  }

  // 初始化操作,使用hiprand为SB随机初始化赋值,利用SB乘以SB的装置得到一个对称但不一定正定矩阵，所以在SB*SB转置之前要对SB做特殊操作，例如SB+αI（α是正数），
  // hiprandGenerator_t gen; 可以用一个hiprandGenerator_t随机初始化两个矩阵
  CHECK_Curand(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  CHECK_Curand(hiprandSetPseudoRandomGeneratorSeed(gen, 4321ULL));
  CHECK_Curand(hiprandGenerateUniform(gen, SB, INPUTN * INPUTN));

  // SB+=I，提高SB的秩，一定程度上确保SB为满秩矩阵
  float *h_SB = new float[INPUTN * INPUTN];
  CHECK_Runtime(hipMemcpy(h_SB, SB, sizeof(float) * INPUTN * INPUTN,
                           hipMemcpyDeviceToHost));
  // std::cout << "the SB is:" << std::endl;
  // print_matrix_colmajor(h_SB, INPUTN, INPUTN);
  for (int i{0}; i < INPUTN; ++i) {
    h_SB[i * INPUTN + i] += 1.0f;
  }

  // 看下SB提高秩之后
  // std::cout << "看下SB提高秩之后" << std::endl;
  // print_matrix_colmajor(h_SB, INPUTN, INPUTN);

  CHECK_Runtime(hipMemcpy(SB, h_SB, sizeof(float) * INPUTN * INPUTN,
                           hipMemcpyHostToDevice));

  // A=SB^T*SB
  CHECK_Cublas(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, INPUTN,
                           INPUTN, INPUTN, &Salpha, SB, INPUTN, SB, INPUTN,
                           &Sbeta, SA, INPUTN));

  CHECK_Runtime(hipMemcpy(stemp, SA, sizeof(float) * INPUTN * INPUTN,
                           hipMemcpyDeviceToHost));
  // std::cout << "让SB对称正交化:" << std::endl;
  // print_matrix_colmajor(stemp, INPUTN, INPUTN);

  // 开局先热身
  for (size_t i{0}; i < warmups / 2; ++i) {
    CHECK_Cublas(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, INPUTN,
                             INPUTN, INPUTN, &Salpha, SB, INPUTN, SB, INPUTN,
                             &Sbeta, SB, INPUTN));
  }

  // 要为A分配内存，但是不必初始化，分配内存主要是因为buffer函数需要一个指针参数，并且buffer函数会以此与lda、n做验证，从而确定需要的Lwork
  CHECK_Cusolver(hipsolverDnSpotrf_bufferSize(
      cusolver_handle, HIPBLAS_FILL_MODE_LOWER, INPUTN, SA, INPUTN, &SLwork));

  CHECK_Runtime(hipMalloc((void **)&SWorkspace, sizeof(float) * SLwork));
  if (SWorkspace == nullptr) {
    fprintf(stderr, "Memory allocation failed for SWorkspace\n");
    exit(EXIT_FAILURE);
  }

  CHECK_Runtime(hipMalloc((void **)&(SdevInfo), sizeof(int)));
  float *h_SB_origin;
  h_SB_origin = (float *)malloc(sizeof(float) * INPUTN * INPUTN);
  CHECK_Runtime(hipMemcpy(h_SB_origin, SA, sizeof(float) * INPUTN * INPUTN,
                           hipMemcpyDeviceToHost));

  // 对start进行record
  CHECK_Runtime(hipEventRecord(start, stream));
  // 要为A分配内存并且对A进行初始化
  // 为了精确，所以repeat
  // for (size_t i{0}; i < num_repeats; i++) {
  CHECK_Cusolver(hipsolverDnSpotrf(cusolver_handle, HIPBLAS_FILL_MODE_LOWER,
                                  INPUTN, SA, INPUTN, SWorkspace, SLwork,
                                  SdevInfo));
  // }
  // 对stop进行record
  CHECK_Runtime(hipEventRecord(stop, stream));
  // 同步
  CHECK_Runtime(hipEventSynchronize(stop));
  CHECK_Runtime(hipStreamSynchronize(stream));

  // 作差求elapse
  float SelapsedTime;
  CHECK_Runtime(hipEventElapsedTime(&SelapsedTime, start, stop));
  // 计算latency
  float const Slatency{SelapsedTime};
  // 计算TFLOPS
  float const Stflops{(one_three * INPUTN * INPUTN * INPUTN) /
                      ((Slatency * 1e-3f) * 1e12f)};
  std::cout << "*************************************************" << std::endl;

  std::cout << "开始 " << argv[1] << " " << "x" << " " << argv[1]
            << " 规模的 Single Cholesky 分解" << std::endl;
  std::cout << "*************************************************" << std::endl;

  // 输出TFLOPS
  std::cout << "单精度Cholesky分解的TFLOPS:" << Stflops << " TFLOPS"
            << std::endl;
  // 输出elapse
  printf("\n单精度Cholesky分解的执行时间: %fms   %fs\n", SelapsedTime,
         SelapsedTime / 1000);
  std::cout << "*************************************************" << std::endl;

  // 查看一下SA和h2d_SB

  CHECK_Runtime(hipMemcpy(stemp, SA, sizeof(float) * INPUTN * INPUTN,
                           hipMemcpyDeviceToHost));

  // std::cout << "对single验证之前查看一下输入矩阵SA" << std::endl;
  // print_matrix_colmajor(stemp, INPUTN, INPUTN);

  CHECK_Runtime(hipMemcpy(stemp, h_SB_origin, sizeof(float) * INPUTN * INPUTN,
                           hipMemcpyHostToHost));
  // std::cout << "对single验证之前查看一下输入矩阵h2d_SB" << std::endl;
  // print_matrix_colmajor(stemp, INPUTN, INPUTN);

  float Sresult;
  // 这里传进去的origin要直接放到memcpy里面作为src以device2host的方式，所以这里的origin需要一个device的版本，但是h_DB是host的，所以一直会报错，解决方案：memcpy
  float *h2d_SB;
  CHECK_Runtime(hipMalloc((void **)&h2d_SB, sizeof(float) * INPUTN * INPUTN));
  CHECK_Runtime(hipMemcpy(h2d_SB, h_SB_origin, sizeof(float) * INPUTN * INPUTN,
                           hipMemcpyHostToDevice));
  validate_ss_S(cublas_handle, SA, &Sresult, h2d_SB, INPUTN);

  std::cout << std::fixed << std::setprecision(20);
  std::cout << "the** Spotrf validate value is:" << Sresult << std::endl;

  // clean

  free(h_DB);
  free(h_SB);

  hipsolverDnDestroy(cusolver_handle);
  hipblasDestroy(cublas_handle);
  hipStreamDestroy(stream);
  hipFree(DA);
  hipFree(DB);
  hipFree(DWorkspace);
  hipFree(SA);
  hipFree(SB);
  hipFree(SWorkspace);

  return EXIT_SUCCESS;
}

// validate from ss
int validate_ss_D(hipblasHandle_t handle, double const *matrix, double *result,
                  double const *origin, int N) {

  double alpha = -1.0;
  double beta = 1.0;
  // double *numerator; // 分子
  double nresult;
  // double *denominator; // 分母
  double *temp;
  double dresult;
  double *L;
  double *H_L;
  double *H_matrix;
  double *temp_origin;

  CHECK_Runtime(hipMalloc((void **)&(L), sizeof(double) * N * N));
  CHECK_Runtime(hipMalloc((void **)&(temp), sizeof(double) * N * N));
  CHECK_Runtime(hipMalloc((void **)&(temp_origin), sizeof(double) * N * N));

  // 显示检查最后一个是否出问题
  hipError_t err1 = hipGetLastError();
  std::cout << "the err1 is:" << err1 << std::endl;

  CHECK_Runtime(hipMemcpy(temp_origin, origin, sizeof(double) * N * N,
                           hipMemcpyDeviceToDevice));
  CHECK_Runtime(hipMemcpy(temp, matrix, sizeof(double) * N * N,
                           hipMemcpyDeviceToDevice));

  H_matrix = (double *)malloc(sizeof(double) * N * N);
  H_L = (double *)malloc(sizeof(double) * N * N);
  CHECK_Runtime(hipMemcpy(H_matrix, matrix, sizeof(double) * N * N,
                           hipMemcpyDeviceToHost));
  // std::cout << "the matrix is :" << std::endl;
  // print_matrix_colmajor(H_matrix, N, N);

  // 下三角
  for (int j = 0; j < N; ++j) { // 遍历列
    // 复制下三角部分的元素，包括对角线
    for (int i = j; i < N; ++i) { // 从对角线开始遍历行
      H_L[i + j * N] = H_matrix[i + j * N];
    }
    // 可选地，将上三角部分（对角线之上）的元素设为零
    for (int i = 0; i < j; ++i) { // 遍历对角线之上的行
      H_L[i + j * N] = 0.0;
    }
  }

  // // 上三角
  // //  H_matrix 和 H_U 是指向 n x n 矩阵的指针，矩阵以列主序存储
  // //  将 H_U 初始化为零（可选）
  // for (int j = 0; j < SIZE; ++j) { // 遍历列
  //   // 复制上三角部分的元素，包括对角线
  //   for (int i = 0; i <= j; ++i) { // 遍历行直到对角线
  //     H_L[i + j * SIZE] = H_matrix[i + j * SIZE];
  //   }
  //   // 可选地，将下三角部分（对角线之下）的元素设为零
  //   for (int i = j + 1; i < SIZE; ++i) { // 遍历对角线之下的行
  //     H_L[i + j * SIZE] = 0.0;
  //   }
  // }

  // std::cout << "the L is :" << std::endl;
  // print_matrix_colmajor(H_L, N, N);
  CHECK_Runtime(
      hipMemcpy(L, H_L, sizeof(double) * N * N, hipMemcpyHostToDevice));
  // 上来就算分母A的L2范数
  CHECK_Cublas(hipblasDnrm2(handle, N * N, temp_origin, 1, &dresult));
  std::cout << "the dresult is:" << dresult << std::endl;
  // 在syrk之前查看一下temp_origin的模样
  // std::cout << "before syrk the temp_origin is:" << std::endl;
  double *lookthetemporigin;
  lookthetemporigin = (double *)malloc(sizeof(double) * N * N);
  CHECK_Runtime(hipMemcpy(lookthetemporigin, temp_origin,
                           sizeof(double) * N * N, hipMemcpyDeviceToHost));

  // print_matrix_colmajor(lookthetemporigin, N, N);
  // 用syrk算出A-LL^T
  CHECK_Cublas(hipblasDsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, N, N,
                           &alpha, L, N, &beta, temp_origin, N));
  // syrk算出来的结果是对称的，所以syrk只会按照uplo的指示将结果存于下三角或者是上三角，才意识到，所以还需要对其进行一个映射操作，之后再进行nrm2运算

  std::cout << "after syrk the temp_origin is:" << std::endl;
  CHECK_Runtime(hipMemcpy(lookthetemporigin, temp_origin,
                           sizeof(double) * N * N, hipMemcpyDeviceToHost));
  // print_matrix_colmajor(lookthetemporigin, N, N);

  // 将下三角映射到上三角，从而构成一个对称矩阵
  for (int i = 0; i < N; ++i) {
    for (int j = i + 1; j < N; ++j) {
      lookthetemporigin[j * N + i] =
          lookthetemporigin[i * N + j]; // 将下三角的值赋给上三角
    }
  }
  // std::cout << "映射情况：" << std::endl;
  // print_matrix_colmajor(lookthetemporigin, N, N);
  double *fenzi;
  CHECK_Runtime(hipMalloc((void **)&(fenzi), sizeof(double) * N * N));
  CHECK_Runtime(hipMemcpy(fenzi, lookthetemporigin, sizeof(double) * N * N,
                           hipMemcpyHostToDevice));
  // 用Dnrm2算分子的L2范数
  CHECK_Cublas(hipblasDnrm2(handle, N * N, fenzi, 1, &nresult));
  std::cout << std::fixed << std::setprecision(20);
  std::cout << "the nresult is:" << nresult << std::endl;

  *result = nresult / dresult;

  CHECK_Runtime(hipFree(L));
  CHECK_Runtime(hipFree(temp));
  CHECK_Runtime(hipFree(temp_origin));
  free(H_L);
  free(H_matrix);

  std::cout << "验证完成" << std::endl;

  return EXIT_SUCCESS;
}

int validate_ss_S(hipblasHandle_t handle, float const *matrix, float *result,
                  float const *origin, int N) {

  float alpha = -1.0;
  float beta = 1.0;
  // double *numerator; // 分子
  float nresult;
  // double *denominator; // 分母
  float *temp;
  float dresult;
  float *L;
  float *H_L;
  float *H_matrix;
  float *temp_origin;

  CHECK_Runtime(hipMalloc((void **)&(L), sizeof(float) * N * N));
  CHECK_Runtime(hipMalloc((void **)&(temp), sizeof(float) * N * N));
  CHECK_Runtime(hipMalloc((void **)&(temp_origin), sizeof(float) * N * N));

  // 显示检查最后一个是否出问题
  hipError_t err1 = hipGetLastError();
  std::cout << "the err1 is:" << err1 << std::endl;

  CHECK_Runtime(hipMemcpy(temp_origin, origin, sizeof(float) * N * N,
                           hipMemcpyDeviceToDevice));
  CHECK_Runtime(hipMemcpy(temp, matrix, sizeof(float) * N * N,
                           hipMemcpyDeviceToDevice));

  H_matrix = (float *)malloc(sizeof(float) * N * N);
  H_L = (float *)malloc(sizeof(float) * N * N);
  CHECK_Runtime(hipMemcpy(H_matrix, matrix, sizeof(float) * N * N,
                           hipMemcpyDeviceToHost));
  // std::cout << "the matrix is :" << std::endl;
  // print_matrix_colmajor(H_matrix, N, N);

  // 下三角
  for (int j = 0; j < N; ++j) { // 遍历列
    // 复制下三角部分的元素，包括对角线
    for (int i = j; i < N; ++i) { // 从对角线开始遍历行
      H_L[i + j * N] = H_matrix[i + j * N];
    }
    // 可选地，将上三角部分（对角线之上）的元素设为零
    for (int i = 0; i < j; ++i) { // 遍历对角线之上的行
      H_L[i + j * N] = 0.0;
    }
  }

  // // 上三角
  // //  H_matrix 和 H_U 是指向 n x n 矩阵的指针，矩阵以列主序存储
  // //  将 H_U 初始化为零（可选）
  // for (int j = 0; j < SIZE; ++j) { // 遍历列
  //   // 复制上三角部分的元素，包括对角线
  //   for (int i = 0; i <= j; ++i) { // 遍历行直到对角线
  //     H_L[i + j * SIZE] = H_matrix[i + j * SIZE];
  //   }
  //   // 可选地，将下三角部分（对角线之下）的元素设为零
  //   for (int i = j + 1; i < SIZE; ++i) { // 遍历对角线之下的行
  //     H_L[i + j * SIZE] = 0.0;
  //   }
  // }

  // std::cout << "the L is :" << std::endl;
  // print_matrix_colmajor(H_L, N, N);
  CHECK_Runtime(
      hipMemcpy(L, H_L, sizeof(float) * N * N, hipMemcpyHostToDevice));
  // 上来就算分母A的L2范数
  CHECK_Cublas(hipblasSnrm2(handle, N * N, temp_origin, 1, &dresult));
  std::cout << "the dresult is:" << dresult << std::endl;
  // 在syrk之前查看一下temp_origin的模样
  // std::cout << "before syrk the temp_origin is:" << std::endl;
  float *lookthetemporigin;
  lookthetemporigin = (float *)malloc(sizeof(float) * N * N);
  CHECK_Runtime(hipMemcpy(lookthetemporigin, temp_origin,
                           sizeof(float) * N * N, hipMemcpyDeviceToHost));

  // print_matrix_colmajor(lookthetemporigin, N, N);
  // 用syrk算出A-LL^T
  CHECK_Cublas(hipblasSsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, N, N,
                           &alpha, L, N, &beta, temp_origin, N));
  // syrk算出来的结果是对称的，所以syrk只会按照uplo的指示将结果存于下三角或者是上三角，才意识到，所以还需要对其进行一个映射操作，之后再进行nrm2运算

  // std::cout << "after syrk the temp_origin is:" << std::endl;
  CHECK_Runtime(hipMemcpy(lookthetemporigin, temp_origin,
                           sizeof(float) * N * N, hipMemcpyDeviceToHost));
  // print_matrix_colmajor(lookthetemporigin, N, N);

  // 将下三角映射到上三角，从而构成一个对称矩阵
  // std::cout << "映射情况：" << std::endl;
  for (int i = 0; i < N; ++i) {
    for (int j = i + 1; j < N; ++j) {
      lookthetemporigin[j * N + i] =
          lookthetemporigin[i * N + j]; // 将下三角的值赋给上三角
    }
  }
  // print_matrix_colmajor(lookthetemporigin, N, N);
  float *fenzi;
  CHECK_Runtime(hipMalloc((void **)&(fenzi), sizeof(float) * N * N));
  CHECK_Runtime(hipMemcpy(fenzi, lookthetemporigin, sizeof(float) * N * N,
                           hipMemcpyHostToDevice));

  // 用Dnrm2算分子的L2范数
  CHECK_Cublas(hipblasSnrm2(handle, N * N, fenzi, 1, &nresult));
  std::cout << std::fixed << std::setprecision(20);
  std::cout << "the nresult is:" << nresult << std::endl;

  *result = nresult / dresult;

  CHECK_Runtime(hipFree(L));
  CHECK_Runtime(hipFree(temp));
  CHECK_Runtime(hipFree(temp_origin));
  free(H_L);
  free(H_matrix);

  return EXIT_SUCCESS;
}

int print_matrix_rowmajor(double *matrix, int m, int n) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      // printf(" %lf", matrix[i * m + j]);
      std::cout << " " << matrix[j * m + i];
    }
    printf("\n");
  }
  return EXIT_SUCCESS;
}

template <typename T> int print_matrix_colmajor(T *matrix, int m, int n) {
  std::cout << std::fixed << std::setprecision(6);

  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < m; ++j) {
      // printf(" %lf", matrix[j * m + i]);
      std::cout << " " << matrix[j * m + i];
    }
    printf("\n");
  }
  return EXIT_SUCCESS;
}
