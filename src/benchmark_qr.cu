#include "../include/cuda_utils_check.hpp"
#include "hipblas.h"
#include <cstddef>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <iostream>
#include <istream>
#include <iterator>
#include <ostream>

int curandSgenerate(float *matrx, int m, int n, unsigned long long seed);
int curandDgenerate(double *matrx, int m, int n, unsigned long long seed);

int main() {

  // 生成handle、各常用参数以及event
  hipsolverHandle_t cusolver_handle;
  CHECK_Cusolver(hipsolverDnCreate(&cusolver_handle));

  float *SA;
  float *SWorkspace;
  double *DA;
  double *DWorkspace;
  int SIZE = 7;
  int SLwork;
  int *SdevInfo;
  float *STAU;
  double *DTAU;
  int DLwork;
  int *DdevInfo;

  hipEvent_t start, stop;
  if (hipEventCreate(&start) != hipSuccess) {
    printf("Failed to create start event\n");
    return EXIT_SUCCESS;
  }

  if (hipEventCreate(&stop) != hipSuccess) {
    printf("Failed to create stop event\n");
    CHECK_Runtime(hipEventDestroy(start));
    return EXIT_SUCCESS;
  }

  CHECK_Runtime(hipMalloc((void **)&SA, sizeof(float) * SIZE * SIZE));
  curandSgenerate(SA, SIZE, SIZE, 1234ULL);
  // Sbuffer_qr
  CHECK_Cusolver(hipsolverDnSgeqrf_bufferSize(cusolver_handle, SIZE, SIZE, SA,
                                             SIZE, &SLwork));

  CHECK_Runtime(hipMalloc((void **)&SWorkspace, SLwork));
  if (SWorkspace == nullptr) {
    fprintf(stderr, "Memory allocation failed for SWorkspace\n");
    exit(EXIT_FAILURE);
  }
  CHECK_Runtime(hipMalloc((void **)&SdevInfo, sizeof(int)));
  if (SdevInfo == nullptr) {
    fprintf(stderr, "Memory allocation failed for SdevInfo\n");
    exit(EXIT_FAILURE);
  }
  CHECK_Runtime(hipMalloc((void **)&STAU, sizeof(float) * SIZE));
  if (STAU == nullptr) {
    fprintf(stderr, "Memory allocation failed for TAU\n");
    exit(EXIT_FAILURE);
  }

  // Sqr
  CHECK_Runtime(hipEventRecord(start));
  CHECK_Cusolver(hipsolverDnSgeqrf(cusolver_handle, SIZE, SIZE, SA, SIZE, STAU,
                                  SWorkspace, SLwork, SdevInfo));
  CHECK_Runtime(hipEventRecord(stop));
  CHECK_Runtime(hipEventSynchronize(stop));
  // 作差求elapse
  float SelapsedTime;
  CHECK_Runtime(hipEventElapsedTime(&SelapsedTime, start, stop));
  // 输出elapse
  printf("\n DnSgeqrf execution time: %fms   %fs\n", SelapsedTime,
         SelapsedTime / 1000);

  CHECK_Runtime(hipMalloc((void **)&DA, sizeof(float) * SIZE * SIZE));
  curandDgenerate(DA, SIZE, SIZE, 1234ULL);
  // Dbuffer_qr
  CHECK_Cusolver(hipsolverDnDgeqrf_bufferSize(cusolver_handle, SIZE, SIZE, DA,
                                             SIZE, &DLwork));

  CHECK_Runtime(hipMalloc((void **)&DWorkspace, DLwork));
  if (DWorkspace == nullptr) {
    fprintf(stderr, "Memory allocation failed for DWorkspace\n");
    exit(EXIT_FAILURE);
  }
  CHECK_Runtime(hipMalloc((void **)&DdevInfo, sizeof(int)));
  if (DdevInfo == nullptr) {
    fprintf(stderr, "Memory allocation failed for DdevInfo\n");
    exit(EXIT_FAILURE);
  }
  CHECK_Runtime(hipMalloc((void **)&DTAU, sizeof(float) * SIZE));
  if (DTAU == nullptr) {
    fprintf(stderr, "Memory allocation failed for DAU\n");
    exit(EXIT_FAILURE);
  }

  // Dqr
  CHECK_Runtime(hipEventRecord(start));
  CHECK_Cusolver(hipsolverDnDgeqrf(cusolver_handle, SIZE, SIZE, DA, SIZE, DTAU,
                                  DWorkspace, DLwork, DdevInfo));
  CHECK_Runtime(hipEventRecord(stop));
  CHECK_Runtime(hipEventSynchronize(stop));
  // 作差求elapse
  float DelapsedTime;
  CHECK_Runtime(hipEventElapsedTime(&DelapsedTime, start, stop));
  // 输出elapse
  printf("\n DnDgeqrf execution time: %fms   %fs\n", DelapsedTime,
         DelapsedTime / 1000);
  return EXIT_SUCCESS;
}

// 生成Double随机数
int curandDgenerate(double *matrx, int m, int n, unsigned long long seed) {
  hiprandGenerator_t gen;
  size_t Sum = m * n;

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);
  hiprandGenerateUniformDouble(gen, matrx, Sum);

  return EXIT_SUCCESS;
}
// 生成Single随机数
int curandSgenerate(float *matrx, int m, int n, unsigned long long seed) {
  hiprandGenerator_t gen;
  size_t Sum = m * n;

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);
  hiprandGenerateUniform(gen, matrx, Sum);

  return EXIT_SUCCESS;
}