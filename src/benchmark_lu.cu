#include "../include/cuda_utils_check.hpp"
#include "hipblas.h"
#include <cstddef>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <iostream>
#include <istream>
#include <iterator>
#include <ostream>

int curandSgenerate(float *matrx, int m, int n, unsigned long long seed);
int curandDgenerate(double *matrx, int m, int n, unsigned long long seed);

int main() {

  hipsolverHandle_t cusolver_handle;
  CHECK_Cusolver(hipsolverDnCreate(&cusolver_handle));

  float *SA;
  float *SWorkspace;
  double *DA;
  double *DWorkspace;
  int SIZE = 7;
  int SLwork;
  int *SdevIpiv;
  int *SdevInfo;
  int DLwork;
  int *DdevIpiv;
  int *DdevInfo;

  hipEvent_t start, stop;
  if (hipEventCreate(&start) != hipSuccess) {
    printf("Failed to create start event\n");
    return EXIT_SUCCESS;
  }

  if (hipEventCreate(&stop) != hipSuccess) {
    printf("Failed to create stop event\n");
    CHECK_Runtime(hipEventDestroy(start));
    return EXIT_SUCCESS;
  }

  // Single LU分解
  CHECK_Runtime(hipMalloc((void **)&SA, sizeof(float) * SIZE * SIZE));
  curandSgenerate(SA, SIZE, SIZE, 1234ULL);
  CHECK_Cusolver(hipsolverDnSgetrf_bufferSize(cusolver_handle, SIZE, SIZE, SA,
                                             SIZE, &SLwork));

  CHECK_Runtime(hipMalloc((void **)&SWorkspace, SLwork));
  if (SWorkspace == nullptr) {
    fprintf(stderr, "Memory allocation failed for SWorkspace\n");
    exit(EXIT_FAILURE);
  }
  CHECK_Runtime(hipMalloc((void **)&SdevInfo, sizeof(int)));
  if (SdevInfo == nullptr) {
    fprintf(stderr, "Memory allocation failed for SdevInfo\n");
    exit(EXIT_FAILURE);
  }
  CHECK_Runtime(hipMalloc((void **)&SdevIpiv, sizeof(int)));
  if (SdevIpiv == nullptr) {
    fprintf(stderr, "Memory allocation failed for SdevIpiv\n");
    exit(EXIT_FAILURE);
  }

  CHECK_Runtime(hipEventRecord(start));
  CHECK_Cusolver(hipsolverDnSgetrf(cusolver_handle, SIZE, SIZE, SA, SIZE,
                                  SWorkspace, SdevIpiv, SdevInfo));
  CHECK_Runtime(hipEventRecord(stop));
  CHECK_Runtime(hipEventSynchronize(stop));
  // 作差求elapse
  float SelapsedTime;
  CHECK_Runtime(hipEventElapsedTime(&SelapsedTime, start, stop));
  // 输出elapse
  printf("\n DnSgetrf execution time: %fms   %fs\n", SelapsedTime,
         SelapsedTime / 1000);

  // Double LU分解
  CHECK_Runtime(hipMalloc((void **)&DA, sizeof(double) * SIZE * SIZE));
  curandDgenerate(DA, SIZE, SIZE, 4321ULL);
  CHECK_Cusolver(hipsolverDnDgetrf_bufferSize(cusolver_handle, SIZE, SIZE, DA,
                                             SIZE, &DLwork));

  CHECK_Runtime(hipMalloc((void **)&DWorkspace, DLwork));
  if (DWorkspace == nullptr) {
    fprintf(stderr, "Memory allocation failed for DWorkspace\n");
    exit(EXIT_FAILURE);
  }
  CHECK_Runtime(hipMalloc((void **)&DdevInfo, sizeof(int)));
  if (DdevInfo == nullptr) {
    fprintf(stderr, "Memory allocation failed for DdevInfo\n");
    exit(EXIT_FAILURE);
  }
  CHECK_Runtime(hipMalloc((void **)&DdevIpiv, sizeof(int)));
  if (DdevIpiv == nullptr) {
    fprintf(stderr, "Memory allocation failed for DdevIpiv\n");
    exit(EXIT_FAILURE);
  }

  CHECK_Runtime(hipEventRecord(start));
  CHECK_Cusolver(hipsolverDnDgetrf(cusolver_handle, SIZE, SIZE, DA, SIZE,
                                  DWorkspace, DdevIpiv, DdevInfo));
  CHECK_Runtime(hipEventRecord(stop));
  CHECK_Runtime(hipEventSynchronize(stop));
  // 作差求elapse
  float DelapsedTime;
  CHECK_Runtime(hipEventElapsedTime(&DelapsedTime, start, stop));
  // 输出elapse
  printf("\n DnDgetrf execution time: %fms   %fs\n", DelapsedTime,
         DelapsedTime / 1000);
}

// 生成Double
int curandDgenerate(double *matrx, int m, int n, unsigned long long seed) {
  hiprandGenerator_t gen;
  size_t Sum = m * n;

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);
  hiprandGenerateUniformDouble(gen, matrx, Sum);

  return EXIT_SUCCESS;
}
// 生成Single
int curandSgenerate(float *matrx, int m, int n, unsigned long long seed) {
  hiprandGenerator_t gen;
  size_t Sum = m * n;

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);
  hiprandGenerateUniform(gen, matrx, Sum);

  return EXIT_SUCCESS;
}